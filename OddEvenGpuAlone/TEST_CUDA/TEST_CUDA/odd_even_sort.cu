#include "hip/hip_runtime.h"
#include "odd_even_sort.cuh"
#include <cmath>
#include <functional>
#include <mutex>
#include <thread>
#include <vector>

__global__ void OddEven(int* arr, int length, int phase) {
    int index = 2 * (blockIdx.x * blockDim.x + threadIdx.x) + phase; //get global index
    if (index >= length - 1) return; //check if index is out of bounds

    int current = arr[index];
    int next = arr[index + 1];

    if (current > next)
    {
        arr[index] = next;
        arr[index + 1] = current;
    }
}

int RoundUpToMultiple(float num, int multiple)
{
    return std::ceil(num / (float)multiple) * multiple;
}

void CalculateThreadsBlocksAmount(int& threads, int& blocks, int length)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    const int threadsAmountMin = 32;
    const int blocksPerMultiMax = deviceProp.maxBlocksPerMultiProcessor;
    const int multiMax = deviceProp.multiProcessorCount;

    if (length > multiMax * blocksPerMultiMax * deviceProp.maxThreadsPerBlock)
    {
        throw std::runtime_error("Array is too big");
    }

    blocks = multiMax * blocksPerMultiMax;
    threads = length / (float)blocks < threadsAmountMin ? threadsAmountMin : RoundUpToMultiple(length / (float)blocks, threadsAmountMin);
}

void sorting::GpuOddEvenSort(std::vector<int>& arr)
{
    int* deviceArr;
    hipMalloc(&deviceArr, arr.size() * sizeof(int));
    hipMemcpy(deviceArr, arr.data(), arr.size() * sizeof(int), hipMemcpyHostToDevice);

    int blocks, threads;
    CalculateThreadsBlocksAmount(threads, blocks, arr.size());

    hipStream_t stream;
    hipStreamCreate(&stream);

    for (int i = 0; i < arr.size(); i++)
    {
        OddEven << <blocks, threads, 0, stream >> > (deviceArr, arr.size(), i % 2);
    }
    hipMemcpy(arr.data(), deviceArr, arr.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceArr);
    hipStreamDestroy(stream);
}