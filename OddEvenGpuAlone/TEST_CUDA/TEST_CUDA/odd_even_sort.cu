#include "hip/hip_runtime.h"
#include "odd_even_sort.cuh"
#include <cmath>
#include <functional>
#include <mutex>
#include <thread>
#include <vector>

__global__ void OddEven(int* arr, int length, int phase) {
    int index = 2 * (blockIdx.x * blockDim.x + threadIdx.x) + phase;
    
    if (index + 1 >= length) return;
	int current = arr[index];
	int next = arr[index + 1];
	if (current > next)
	{
		arr[index] = next;
		arr[index + 1] = current;
	}
}

int RoundUpToMultiple(int num, int multiple)
{
    return std::ceil(num / (float)multiple) * multiple;
}

void CalculateThreadsBlocksAmount(int& threads, int& blocks, int length)
{
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    
    const int threadsAmountMin = 32;
    const int blocksPerMultiMax = deviceProp.maxBlocksPerMultiProcessor;
	const int multiMax = deviceProp.multiProcessorCount;

    if (length > multiMax * blocksPerMultiMax * deviceProp.maxThreadsPerBlock)
    {
		throw std::runtime_error("Array is too big");
    }

    blocks = multiMax * blocksPerMultiMax;
	threads = RoundUpToMultiple(length / blocks, threadsAmountMin);
}



void sorting::GpuOddEvenSort(std::vector<int>& arr)
{
    int* deviceArr;
    hipMalloc(&deviceArr, arr.size() * sizeof(int));
    hipMemcpy(deviceArr, arr.data(), arr.size() * sizeof(int), hipMemcpyHostToDevice);

    int blocks, threads;
	CalculateThreadsBlocksAmount(threads, blocks, std::ceill(arr.size() / 2.0));

    hipStream_t stream;
    hipStreamCreate(&stream);

    for (int i = 0; i < arr.size(); i++)
    {
        OddEven << <blocks, threads, 0, stream >> > (deviceArr, arr.size(), i%2);
    }
    hipMemcpy(arr.data(), deviceArr, arr.size() * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(deviceArr);
    hipStreamDestroy(stream);
}
