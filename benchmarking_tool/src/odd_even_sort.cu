#include "hip/hip_runtime.h"
#include "odd_even_sort.cuh"


__global__ void sorting::Even(int* arr, int length) {
	int index = 2 * (blockIdx.x * blockDim.x + threadIdx.x); //get global index
	if (index >= length - 1) return; //check if index is out of bounds

    //compare and swap
	if (arr[index] > arr[index + 1]) 
    { 
        int tmp = arr[index];
        arr[index] = arr[index + 1];
        arr[index + 1] = tmp;
    }
}

__global__ void sorting::Odd(int* arr, int length) {
    int index = 2 * (blockIdx.x * blockDim.x + threadIdx.x) + 1; //get global index
    if (index >= length - 1) return; //check if index is out of bounds

    if (arr[index] > arr[index + 1]) 
    {
        int tmp = arr[index];
        arr[index] = arr[index + 1];
        arr[index + 1] = tmp;
    }
}

void sorting::GpuOddEvenSort(std::vector<int>& arr)
{
	int half = arr.size() / 2; //get half size of the array
    int* d_arr; //arr copy for gpu
	hipMalloc(&d_arr, arr.size() * sizeof(int)); //allocate memory for d_arr
    hipMemcpy(d_arr, arr.data(), arr.size() * sizeof(int), hipMemcpyHostToDevice); //copy

	int threads = 256; //threads per block (should be multiple of 32)

    //number of blocks. 
    //half of array size is used because the odd and even idexes are handled at the same time
	//this calculation guarantees that number of threads is enough to handle all elements
	int blocks = (int)ceil(half / (double)threads); 

	//half iterations because we handle even and odd indexes at the same time
    for (int i = 0; i < half; i++)
    {
        sorting::Even << <blocks, threads >> > (d_arr, arr.size()); //handle even
        sorting::Odd << <blocks, threads >> > (d_arr, arr.size()); //handle odd
		hipDeviceSynchronize(); //wait for all threads to finish
    }
	hipMemcpy(arr.data(), d_arr, arr.size() * sizeof(int), hipMemcpyDeviceToHost); //copy back

	hipFree(d_arr); //free memory
}


void sorting::CpuOddEvenSort(std::vector<int>& arr)
{
    bool sorted = false;
    while (!sorted)
    {
        sorted = true;

        std::thread t1([&arr, &sorted] {
            for (int i = 0; i < arr.size() - 1; i += 2)
            {
                if (arr[i] > arr[i + 1])
                {
                    std::swap(arr[i], arr[i + 1]);
                    sorted = false;
                }
            }
            });

        std::thread t2([&arr, &sorted] {
            for (int i = 1; i < arr.size() - 1; i += 2)
            {
                if (arr[i] > arr[i + 1])
                {
                    std::swap(arr[i], arr[i + 1]);
                    sorted = false;
                }
            }
            });

        if (t1.joinable()) t1.join();
        if (t2.joinable()) t2.join();
    }
}
